#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "stdio.h"
#include <iostream>
#include <hip/hip_fp16.h>
#include "utils.hpp"

struct TransInfo{
    int src_w;
    int src_h;
    int tar_w;
    int tar_h;
    TransInfo(int srcW, int srcH, int tarW, int tarH):
        src_w(srcW), src_h(srcH), tar_w(tarW), tar_h(tarH){}
};

struct AffineMatrix{
    float forward[6];
    float reverse[6];
    float forward_scale;
    float reverse_scale;

    void calc_forward_matrix(TransInfo trans){
        forward[0] = forward_scale;
        forward[1] = 0;
        forward[2] = - forward_scale * trans.src_w * 0.5 + trans.tar_w * 0.5;
        forward[3] = 0;
        forward[4] = forward_scale;
        forward[5] = - forward_scale * trans.src_h * 0.5 + trans.tar_h * 0.5;
    };

    void calc_reverse_matrix(TransInfo trans){
        reverse[0] = reverse_scale;
        reverse[1] = 0;
        reverse[2] = - reverse_scale * trans.tar_w * 0.5 + trans.src_w * 0.5;
        reverse[3] = 0;
        reverse[4] = reverse_scale;
        reverse[5] = - reverse_scale * trans.tar_h * 0.5 + trans.src_h * 0.5;
    };

    void init(TransInfo trans){
        float scaled_w = (float)trans.tar_w / trans.src_w;
        float scaled_h = (float)trans.tar_h / trans.src_h;
        forward_scale = (scaled_w < scaled_h ? scaled_w : scaled_h);
        reverse_scale = 1 / forward_scale;
    
        // 计算src->tar和tar->src的仿射矩阵
        calc_forward_matrix(trans);
        calc_reverse_matrix(trans);
    }
};

__device__ void affine_transformation(
    float* trans_matrix, 
    int src_x, int src_y, 
    float* tar_x, float* tar_y)
{
    *tar_x = trans_matrix[0] * src_x + trans_matrix[1] * src_y + trans_matrix[2];
    *tar_y = trans_matrix[3] * src_x + trans_matrix[4] * src_y + trans_matrix[5];
}


__global__ void resize_nearest_BGR2RGB_kernel(
    uint8_t* tar, uint8_t* src, 
    int tarW, int tarH, 
    int srcW, int srcH,
    float scaled_w, float scaled_h) 
{
    // nearest neighbour -- resized之后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // nearest neighbour -- 计算最近坐标
    int src_y = round((float)y * scaled_h);
    int src_x = round((float)x * scaled_w);

    if (src_x < 0 || src_y < 0 || src_x > srcW || src_y > srcH) {
        // nearest neighbour -- 对于越界的部分，不进行计算
    } else {
        // nearest neighbour -- 计算tar中对应坐标的索引
        int tarIdx = (y * tarW  + x) * 3;

        // nearest neighbour -- 计算src中最近邻坐标的索引
        int srcIdx = (src_y * srcW + src_x) * 3;

        // nearest neighbour -- 实现nearest beighbour的resize + BGR2RGB
        tar[tarIdx + 0] = src[srcIdx + 2];
        tar[tarIdx + 1] = src[srcIdx + 1];
        tar[tarIdx + 2] = src[srcIdx + 0];
    }
}

template <typename T>
__global__ void resize_bilinear_hwc2chw_kernel(
    T* tar, uint8_t* src, 
    int tarW, int tarH, 
    int srcW, int srcH, 
    float scaled_w, float scaled_h,
    T mean, T std) 
{

    // bilinear interpolation -- resized之后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // bilinear interpolation -- 计算x,y映射到原图时最近的4个坐标
    int src_y1 = floor((y + 0.5) * scaled_h - 0.5);
    int src_x1 = floor((x + 0.5) * scaled_w - 0.5);
    int src_y2 = src_y1 + 1;
    int src_x2 = src_x1 + 1;

    if (src_y1 < 0 || src_x1 < 0 || src_y1 > srcH || src_x1 > srcW) {
        // bilinear interpolation -- 对于越界的坐标不进行计算
    } else {
        // bilinear interpolation -- 计算原图上的坐标(浮点类型)在0~1之间的值
        T th   = ((y + 0.5) * scaled_h - 0.5) - src_y1;
        T tw   = ((x + 0.5) * scaled_w - 0.5) - src_x1;

        // bilinear interpolation -- 计算面积(这里建议自己手画一张图来理解一下)
        T a1_1 = (T(1.0) - tw) * (T(1.0) - th);  //右下
        T a1_2 = tw * (T(1.0) - th);          //左下
        T a2_1 = (T(1.0) - tw) * th;          //右上
        T a2_2 = tw * th;                  //左上

        // bilinear interpolation -- 计算4个坐标所对应的索引
        int srcIdx1_1 = (src_y1 * srcW + src_x1) * 3;  //左上
        int srcIdx1_2 = (src_y1 * srcW + src_x2) * 3;  //右上
        int srcIdx2_1 = (src_y2 * srcW + src_x1) * 3;  //左下
        int srcIdx2_2 = (src_y2 * srcW + src_x2) * 3;  //右下

        // bilinear interpolation -- 计算resized之后的图的索引
        int tarIdx    = y * tarW  + x;
        int tarArea   = tarW * tarH;

        // bilinear interpolation -- 实现bilinear interpolation的resize
        // if BGR2RGB a1_1 * src[srcIdx1_1 + 0] ->  a1_1 * src[srcIdx1_1 + 2]
        tar[tarIdx + tarArea * 0] = ((
                          a1_1 * T(src[srcIdx1_1 + 0]) + 
                          a1_2 * T(src[srcIdx1_2 + 0]) +
                          a2_1 * T(src[srcIdx2_1 + 0]) +
                          a2_2 * T(src[srcIdx2_2 + 0])) - mean) * std;

        tar[tarIdx + tarArea * 1] = ((
                          a1_1 * T(src[srcIdx1_1 + 1]) + 
                          a1_2 * T(src[srcIdx1_2 + 1]) +
                          a2_1 * T(src[srcIdx2_1 + 1]) +
                          a2_2 * T(src[srcIdx2_2 + 1])) - mean) * std;

        tar[tarIdx + tarArea * 2] = ((
                          a1_1 * T(src[srcIdx1_1 + 2]) + 
                          a1_2 * T(src[srcIdx1_2 + 2]) +
                          a2_1 * T(src[srcIdx2_1 + 2]) +
                          a2_2 * T(src[srcIdx2_2 + 2])) - mean) * std;
    }
}

__global__ void resize_bilinear_BGR2RGB_shift_kernel(
    uint8_t* tar, uint8_t* src, 
    int tarW, int tarH, 
    int srcW, int srcH, 
    float scaled_w, float scaled_h) 
{
    // resized之后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // bilinear interpolation -- 计算x,y映射到原图时最近的4个坐标
    int src_y1 = floor((y + 0.5) * scaled_h - 0.5);
    int src_x1 = floor((x + 0.5) * scaled_w - 0.5);
    int src_y2 = src_y1 + 1;
    int src_x2 = src_x1 + 1;

    if (src_y1 < 0 || src_x1 < 0 || src_y1 > srcH || src_x1 > srcW) {
        // bilinear interpolation -- 对于越界的坐标不进行计算
    } else {
        // bilinear interpolation -- 计算原图上的坐标(浮点类型)在0~1之间的值
        float th   = ((y + 0.5) * scaled_h - 0.5) - src_y1;
        float tw   = ((x + 0.5) * scaled_w - 0.5) - src_x1;

        // bilinear interpolation -- 计算面积(这里建议自己手画一张图来理解一下)
        float a1_1 = (1.0 - tw) * (1.0 - th);  //右下
        float a1_2 = tw * (1.0 - th);          //左下
        float a2_1 = (1.0 - tw) * th;          //右上
        float a2_2 = tw * th;                  //左上

        // bilinear interpolation -- 计算4个坐标所对应的索引
        int srcIdx1_1 = (src_y1 * srcW + src_x1) * 3;  //左上
        int srcIdx1_2 = (src_y1 * srcW + src_x2) * 3;  //右上
        int srcIdx2_1 = (src_y2 * srcW + src_x1) * 3;  //左下
        int srcIdx2_2 = (src_y2 * srcW + src_x2) * 3;  //右下

        // bilinear interpolation -- 计算原图在目标图中的x, y方向上的偏移量
        y = y - int(srcH / (scaled_h * 2)) + int(tarH / 2);
        x = x - int(srcW / (scaled_w * 2)) + int(tarW / 2);

        // bilinear interpolation -- 计算resized之后的图的索引
        int tarIdx    = (y * tarW  + x) * 3;

        // bilinear interpolation -- 实现bilinear interpolation + BGR2RGB
        tar[tarIdx + 0] = round(
                          a1_1 * src[srcIdx1_1 + 2] + 
                          a1_2 * src[srcIdx1_2 + 2] +
                          a2_1 * src[srcIdx2_1 + 2] +
                          a2_2 * src[srcIdx2_2 + 2]);

        tar[tarIdx + 1] = round(
                          a1_1 * src[srcIdx1_1 + 1] + 
                          a1_2 * src[srcIdx1_2 + 1] +
                          a2_1 * src[srcIdx2_1 + 1] +
                          a2_2 * src[srcIdx2_2 + 1]);

        tar[tarIdx + 2] = round(
                          a1_1 * src[srcIdx1_1 + 0] + 
                          a1_2 * src[srcIdx1_2 + 0] +
                          a2_1 * src[srcIdx2_1 + 0] +
                          a2_2 * src[srcIdx2_2 + 0]);
    }
}

__global__ void resize_warpaffine_BGR2RGB_kernel(
    uint8_t* tar, uint8_t* src, 
    TransInfo trans,
    AffineMatrix matrix)
{
    float src_x, src_y;

    // resized之后的图tar上的坐标
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // bilinear interpolation -- 通过逆仿射变换得到计算tar中的x, y所需要的src中的src_x, src_y
    affine_transformation(matrix.reverse, x + 0.5, y + 0.5, &src_x, &src_y);

    // bilinear interpolation -- 计算x,y映射到原图时最近的4个坐标
    int src_x1 = floor(src_x - 0.5);
    int src_y1 = floor(src_y - 0.5);
    int src_x2 = src_x1 + 1;
    int src_y2 = src_y1 + 1;

    if (src_y1 < 0 || src_x1 < 0 || src_y1 > trans.src_h || src_x1 > trans.src_w) {
        // bilinear interpolation -- 对于越界的坐标不进行计算
    } else {
        // bilinear interpolation -- 计算原图上的坐标(浮点类型)在0~1之间的值
        float tw   = src_x - src_x1;
        float th   = src_y - src_y1;

        // bilinear interpolation -- 计算面积(这里建议自己手画一张图来理解一下)
        float a1_1 = (1.0 - tw) * (1.0 - th);  //右下
        float a1_2 = tw * (1.0 - th);          //左下
        float a2_1 = (1.0 - tw) * th;          //右上
        float a2_2 = tw * th;                  //左上

        // bilinear interpolation -- 计算4个坐标所对应的索引
        int srcIdx1_1 = (src_y1 * trans.src_w + src_x1) * 3;  //左上
        int srcIdx1_2 = (src_y1 * trans.src_w + src_x2) * 3;  //右上
        int srcIdx2_1 = (src_y2 * trans.src_w + src_x1) * 3;  //左下
        int srcIdx2_2 = (src_y2 * trans.src_w + src_x2) * 3;  //右下


        // bilinear interpolation -- 计算resized之后的图的索引
        int tarIdx    = (y * trans.tar_w  + x) * 3;

        // bilinear interpolation -- 实现bilinear interpolation + BGR2RGB
        tar[tarIdx + 0] = round(
                          a1_1 * src[srcIdx1_1 + 2] + 
                          a1_2 * src[srcIdx1_2 + 2] +
                          a2_1 * src[srcIdx2_1 + 2] +
                          a2_2 * src[srcIdx2_2 + 2]);

        tar[tarIdx + 1] = round(
                          a1_1 * src[srcIdx1_1 + 1] + 
                          a1_2 * src[srcIdx1_2 + 1] +
                          a2_1 * src[srcIdx2_1 + 1] +
                          a2_2 * src[srcIdx2_2 + 1]);

        tar[tarIdx + 2] = round(
                          a1_1 * src[srcIdx1_1 + 0] + 
                          a1_2 * src[srcIdx1_2 + 0] +
                          a2_1 * src[srcIdx2_1 + 0] +
                          a2_2 * src[srcIdx2_2 + 0]);
    }
}

/*
    这里面的所有函数都实现了kernel fusion。这样可以减少kernel launch所产生的overhead
    如果使用了shared memory的话，就可以减少分配shared memory所产生的overhead以及内部线程同步的overhead。(这个案例没有使用shared memory)
    CUDA编程中有一些cuda runtime api是implicit synchronize(隐式同步)的，比如hipMalloc, hipHostMalloc，以及shared memory的分配。
    高效的CUDA编程需要意识这些implicit synchronize以及其他会产生overhead的地方。比如使用内存复用的方法，让cuda分配完一次memory就一直使用它

    这里建议大家把我写的每一个kernel都拆开成不同的kernel来分别计算
    e.g. resize kernel + BGR2RGB kernel + shift kernel 
    之后用nsight去比较融合与不融合的差别在哪里。去体会一下fusion的好处
*/


template <typename T>
void resize_bilinear_gpu(
    T* d_tar, uint8_t* d_src, 
    int tarW, int tarH, 
    int srcW, int srcH, 
    int tactis, T mean, T std) 
{
    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid(tarW / 16 + 1, tarH / 16 + 1, 1);
    
    //scaled resize
    float scaled_h = (float)srcH / tarH;
    float scaled_w = (float)srcW / tarW;
    float scale = (scaled_h > scaled_w ? scaled_h : scaled_w);

    if (tactis > 1) {
        scaled_h = scale;
        scaled_w = scale;
    }

    // for affine transformation
    TransInfo    trans(srcW, srcH, tarW, tarH);
    AffineMatrix affine;
    affine.init(trans);
    
    resize_bilinear_hwc2chw_kernel<T><<<dimGrid, dimBlock>>> (d_tar, d_src, tarW, tarH, srcW, srcH, scaled_w, scaled_h, mean, std);
}


template __global__ void resize_bilinear_hwc2chw_kernel<int8_t>(int8_t* tar, uint8_t* src, int tarW, int tarH, int srcW, int srcH, float scaled_w, float scaled_h, int8_t mean, int8_t std);
template __global__ void resize_bilinear_hwc2chw_kernel<half>(half* tar, uint8_t* src, int tarW, int tarH, int srcW, int srcH, float scaled_w, float scaled_h, half mean, half std);
template __global__ void resize_bilinear_hwc2chw_kernel<float>(float* tar, uint8_t* src, int tarW, int tarH, int srcW, int srcH, float scaled_w, float scaled_h, float mean, float std);

// 这里不应该有uint8的模板函数，有均值和方差最少是half 或者仅有mean 可以是int8(mean=128)
template void resize_bilinear_gpu<int8_t>(int8_t* d_tar, uint8_t* d_src, int tarW, int tarH, int srcW, int srcH, int tactis, int8_t mean, int8_t std);
template void resize_bilinear_gpu<half>(half* d_tar, uint8_t* d_src, int tarW, int tarH, int srcW, int srcH, int tactis, half mean, half std);
template void resize_bilinear_gpu<float>(float* d_tar, uint8_t* d_src, int tarW, int tarH, int srcW, int srcH, int tactis, float mean, float std);
